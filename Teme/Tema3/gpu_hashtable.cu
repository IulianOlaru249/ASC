#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

#define hash_func	hashShift

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size)
{
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable()
{
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape)
{
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys)
{
	return false;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys)
{
	return NULL;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor()
{
	return 0.f; // no larger than 1.0f = 100%
}


int GpuHashTable::hashShift(int key)
{
	key = ~key + (key << 15); // key = (key << 15) - key - 1;
	key = key ^ (key >> 12);
	key = key + (key << 2);
	key = key ^ (key >> 4);
	key = key * 2057; // key = (key + (key << 3)) + (key << 11);
	key = key ^ (key >> 16);

	return key;
}

int GpuHashTable::hashJenkins(int key)
{
	key = (key + 0x7ed55d16) + (key << 12);
	key = (key ^ 0xc761c23c) ^ (key >> 19);
	key = (key + 0x165667b1) + (key << 5);
	key = (key + 0xd3a2646c) ^ (key << 9);
	key = (key + 0xfd7046c5) + (key << 3);
	key = (key ^ 0xb55a4f09) ^ (key >> 16);

	return key;
}

int GpuHashTable::hashShiftMult(int key)
{
	key = (key ^ 61) ^ (key >> 16);
	key = key + (key << 3);
	key = key ^ (key >> 4);
	key = key * 0x27d4eb2d;
	key = key ^ (key >> 15);

	return key;
}

int GpuHashTable::hashKnuth(int key)
{
	return key * 2654435761 % (1LLU << 32);
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
