#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

#define MAX_LOAD_FACTOR		.9f
#define MIN_LOAD_FACTOR		.85f

/* INIT HASH
 */
__device__ size_t computeHash(int key)
{
	size_t hash = (size_t)key;

	hash = ~hash + (hash << 15); // key = (key << 15) - key - 1;
	hash = hash ^ (hash >> 12);
	hash = hash + (hash << 2);
	hash = hash ^ (hash >> 4);
	hash = hash + (hash << 3) + (hash << 11);
	hash = hash ^ (hash >> 16);

	return hash;
}

__global__ void kernel_insert(Entry* hashMap, int* devKeys, int* devValues,
	size_t capacity)
{
	int oldKey;
	bool inserted = false;
	size_t hash;
	Entry insertedEntry;
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > capacity)
	{
		return;
	}

	hash = computeHash(devKeys[idx]) % capacity;
	insertedEntry = {devKeys[idx], devValues[idx]};

	for (; !inserted; hash = (hash + 1) % capacity)
	{
		oldKey = atomicCAS(&hashMap[hash].key, KEY_INVALID, insertedEntry.key);

		if (KEY_INVALID == oldKey || insertedEntry.key == oldKey)
		{
			hashMap[hash].value = insertedEntry.value;
			inserted = true;
		}
	}
}

__global__ void kernel_search(Entry* hashMap, int* devKeys, int* values,
	size_t capacity, int numKeys)
{
	bool found = false;
	size_t hash;
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > numKeys)
	{
		return;
	}

	hash = computeHash(devKeys[idx]) % capacity;

	for (; !found; hash = (hash + 1) % capacity)
	{
		if (devKeys[idx] == hashMap[hash].key)
		{
			values[idx] = hashMap[hash].value;
			found = true;
		}
	}
}

__global__ void kernel_rehash(Entry* resizedHashMap, Entry* hashMap,
	size_t initialCapacity, int finalCapacity)
{
	bool reinserted = false;
	size_t hash;
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > initialCapacity || KEY_INVALID == hashMap[idx].key)
	{
		return;
	}

	hash = computeHash(hashMap[idx].key) % finalCapacity;

	for (; !reinserted; hash = (hash + 1) % finalCapacity)
	{
		if (KEY_INVALID ==
			atomicCAS(&resizedHashMap[hash].key, KEY_INVALID, hashMap[idx].key))
		{
			resizedHashMap[hash].value = hashMap[idx].value;
			reinserted = true;
		}
	}
}

GpuHashTable::GpuHashTable(int size) :
	capacity(size), size(0)
{
	hipError_t ret;
	size_t numBytes = capacity * sizeof(*hashMap);

	ret = hipMalloc(&hashMap, numBytes);
	ASSERT(ret, "hipMalloc(hashMap) failed", exit(errno));

	// O pozitie din hashtable nu contine un element daca cheia de pe aceasta
	// este 0
	ret = hipMemset(hashMap, 0, numBytes);
	ASSERT(ret, "hipMemset(hashMap) failed", exit(errno));
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable()
{
	hipFree(hashMap);
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape)
{
	hipError_t ret;
	int numBlocks;
	int numThreads;
	Entry* resizedHashMap;
	size_t numBytes = numBucketsReshape * sizeof(*resizedHashMap);

	ret = hipMalloc(&resizedHashMap, numBytes);
	ASSERT(ret, "hipMalloc(hashMap) failed", exit(errno));

	ret = hipMemset(resizedHashMap, 0, numBytes);
	ASSERT(ret, "hipMemset(resizedHashMap) failed", exit(EXIT_FAILURE));

	ret = getNumBlocksThreads(numBlocks, numThreads, capacity);
	ASSERT(ret, "getNumBlocksThreads() failed", exit(EXIT_FAILURE));

	kernel_rehash<<<numBlocks, numThreads>>>(resizedHashMap, hashMap,
		capacity, numBucketsReshape);

	ret = hipDeviceSynchronize();
	ASSERT(ret, "hipDeviceSynchronize() failed", exit(EXIT_FAILURE));

	ret = hipFree(hashMap);
	ASSERT(ret, "hipFree(hashMap) failed", exit(EXIT_FAILURE));

	hashMap = resizedHashMap;
	capacity = numBucketsReshape;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int* keys, int* values, int numKeys)
{
	hipError_t ret;
	int numBlocks;
	int numThreads;
	int* devKeys;
	int* devValues;
	size_t numBytes = numKeys * sizeof(*devKeys);

	ret = hipMalloc(&devKeys, numBytes);
	ASSERT(ret, "hipMalloc(devKeys) failed", return false);

	ret = hipMemcpy(devKeys, keys, numBytes, hipMemcpyHostToDevice);
	ASSERT(ret, "hipMemcpy(devKeys) failed", return false);

	ret = hipMalloc(&devValues, numBytes);
	ASSERT(ret, "hipMalloc(devValues) failed", return false);

	ret = hipMemcpy(devValues, values, numBytes, hipMemcpyHostToDevice);
	ASSERT(ret, "hipMemcpy(devValues) failed", return false);

	if ((size + numKeys) / float(capacity) >= MAX_LOAD_FACTOR)
	{
		reshape((size + numKeys) / MIN_LOAD_FACTOR + 1);
	}

	ret = getNumBlocksThreads(numBlocks, numThreads, numKeys);
	ASSERT(ret, "getNumBlocksThreads() failed", return false);

	kernel_insert<<<numBlocks, numThreads>>>(hashMap, devKeys, devValues,
		capacity);

	ret = hipDeviceSynchronize();
	ASSERT(ret, "hipDeviceSynchronize() failed", return false);

	size += numKeys;

	ret = hipFree(devKeys);
	ASSERT(ret, "hipFree(devKeys) failed", return false);

	ret = hipFree(devValues);
	ASSERT(ret, "hipFree(devValues) failed", return false);

	return true;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys)
{
	hipError_t ret;
	int numBlocks;
	int numThreads;
	int* devKeys;
	int* values;
	size_t numBytes = numKeys * sizeof(*devKeys);

	ret = hipMalloc(&devKeys, numBytes);
	ASSERT(ret, "hipMalloc(devKeys) failed", return NULL);

	ret = hipMemcpy(devKeys, keys, numBytes, hipMemcpyHostToDevice);
	ASSERT(ret, "hipMemcpy(devKeys) failed", return NULL);

	ret = hipMallocManaged(&values, numBytes);
	ASSERT(ret, "hipMallocManaged(values) failed", return NULL);

	ret = getNumBlocksThreads(numBlocks, numThreads, numKeys);
	ASSERT(ret, "getNumBlocksThreads() failed", return NULL);

	kernel_search<<<numBlocks, numThreads>>>(hashMap, devKeys, values,
		capacity, numKeys);

	hipDeviceSynchronize();

	ret = hipFree(devKeys);
	ASSERT(ret, "hipMalloc(devKeys) failed",);

	return values;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor()
{
	return (float)size / capacity; // no larger than 1.0f = 100%
}

hipError_t GpuHashTable::getNumBlocksThreads(int& numBlocks, int& numThreads,
	int numKeys)
{
	hipError_t ret;
	hipDeviceProp_t devProp;

	ret = hipGetDeviceProperties(&devProp, 0);
	ASSERT(ret, "hipGetDeviceProperties failed", return ret);

	// fprintf(stderr, "threads = %d\n", devProp.maxThreadsPerBlock);
	numThreads = devProp.maxThreadsPerBlock;
	numBlocks = numKeys / numThreads;

	if (numBlocks * numThreads != numKeys)
	{
		++numBlocks;
	}

	return hipSuccess;
}

int GpuHashTable::hashShift(int key)
{
	key = ~key + (key << 15); // key = (key << 15) - key - 1;
	key = key ^ (key >> 12);
	key = key + (key << 2);
	key = key ^ (key >> 4);
	key = key * 2057; // key = (key + (key << 3)) + (key << 11);
	key = key ^ (key >> 16);

	return key;
}

int GpuHashTable::hashJenkins(int key)
{
	key = (key + 0x7ed55d16) + (key << 12);
	key = (key ^ 0xc761c23c) ^ (key >> 19);
	key = (key + 0x165667b1) + (key << 5);
	key = (key + 0xd3a2646c) ^ (key << 9);
	key = (key + 0xfd7046c5) + (key << 3);
	key = (key ^ 0xb55a4f09) ^ (key >> 16);

	return key;
}

int GpuHashTable::hashShiftMult(int key)
{
	key = (key ^ 61) ^ (key >> 16);
	key = key + (key << 3);
	key = key ^ (key >> 4);
	key = key * 0x27d4eb2d;
	key = key ^ (key >> 15);

	return key;
}

int GpuHashTable::hashKnuth(int key)
{
	return key * 2654435761 % (1LLU << 32);
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
